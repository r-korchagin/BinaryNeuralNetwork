#include "hip/hip_runtime.h"
extern "C"

// int DAT_Size           // int MAX index of DAT
// int FAMILY_Size     // int	SIZE Family
// long long *DAT        // long[numberDAT][indexInDAT]
// int DAT_Lenght       // int
// int level_1               // int   size first level
// int level_2
// int level_3
// int level_4
// long long *link1_2    // long[indexFamily][index]
// long long *link2_3    // long[indexFamily][index]
// long long *link3_4    // long[indexFamily][index]

__global__ void neural(
				int DAT_Size,
				int DAT_Lenght,
				int FAMILY_Size,
				long long *DAT,
				int level_1,
				int level_2,
				int level_3,
				int level_4,
				long long *link1_2,
				long long *link2_3,
				long long *link3_4,
				long long *result)
{	
	int j = blockIdx.x * blockDim.x * blockDim.y + threadIdx.y * blockDim.x + threadIdx.x;
	//int idx = threadIdx.x + (((gridDim.x * blockIdx.y) + blockIdx.x)*blockDim.x);
	//int j = blockIdx.x * blockDim.x + threadIdx.x; //thread index	
	if (j<FAMILY_Size * DAT_Size){
	
		int NumberFamily = (int)(j - (FAMILY_Size * ((int)j/FAMILY_Size)));
		int NumberDAT = (int)(j/FAMILY_Size);
		
		long long datlev2[@second_lev@]; //level_2
		long long datlev3[@tride_lev@]; //level_3
		long long datlev4[1]; //level_4
		long long startbit = 1;
		
			// **************  LEVEL 1 - 2   ************************ //			
			int m = 0; // num link 1 - 2
			int currlev[64];
			long long addData;
			long long middleval = 0;
			
			for (int k = 0; k < level_2; ++k) {							
				for (int i = 0; i < DAT_Lenght; ++i){
						int DAT_index = i+NumberDAT*DAT_Lenght;// array2Index(i, NumberDAT, DAT_Lenght); //return x + y*height;
						int link1_2_index = m+NumberFamily*level_1*level_2; // array2Index(m, NumberFamily, level_1); //return x + y*height;
						if ( DAT[DAT_index] != 0) {
							addData = DAT[DAT_index]&link1_2[link1_2_index];
							for (int f = 0; f < 64; ++f){					
								currlev[f] = currlev[f] + (int)((addData>>f)&1);
							}						
							++m;
						}
					}
					int lev = 0;
					for (int f = 0; f < 64; ++f){
						if (currlev[f] > lev){
							lev = currlev[f];
							}
					}
					lev = lev >> 1;
					for (int f = 0; f < 64; ++f){
						if (currlev[f] >= lev){datlev2[k] = datlev2[k]^(startbit<<f);}
					}
					//if(middleval == 0){ middleval = datlev2[k];}
					for (int f = 0; f < 64; ++f){
						currlev[f] = 0;
					}
			}			
			// **************  END LEVEL 1 - 2   ************************ //
			// **************  LEVEL 2 - 3   ************************ //
			m = 0; // num link 2 - 3
			for (int k = 0; k < level_3; ++k) {							
				for (int i = 0; i < level_2; ++i){
						int link2_3_index = m+NumberFamily*level_2*level_3; // array2Index(m, NumberFamily, level_1); //return x + y*height;
						addData = datlev2[i]&link2_3[link2_3_index];
						for (int f = 0; f < 64; ++f){	
							currlev[f] = currlev[f] + (int)((addData>>f)&1);
						}						
						++m;
					}
					
					int lev = 0;
					for (int f = 0; f < 64; ++f){
						if (currlev[f] > lev){
							lev = currlev[f];
							}
					}
					lev = lev >> 1;
					for (int f = 0; f < 64; ++f){
						if (currlev[f] >= lev){datlev3[k] = datlev3[k]^(startbit<<f);}
					}
					for (int f = 0; f < 64; ++f){
						currlev[f] = 0;
					}
			}
			// **************  END LEVEL 2 - 3   ************************ //
			// **************  LEVEL 3 - 4   ************************ //
			m = 0; // num link 3 - 4
			for (int k = 0; k < level_4; ++k) {							
				for (int i = 0; i < level_3; ++i){
						int link3_4_index = m+NumberFamily*level_3*level_4; // array2Index(m, NumberFamily, level_1); //return x + y*height;
						addData = datlev3[i]&link3_4[link3_4_index];
						for (int f = 0; f < 64; ++f){	
							currlev[f] = currlev[f] + (int)((addData>>f)&1);
						}						
						++m;
					}
					
					int lev = 0;
					for (int f = 0; f < 64; ++f){
						if (currlev[f] > lev){
							lev = currlev[f];
							}
					}
					lev = lev >> 1;
					for (int f = 0; f < 64; ++f){
						if (currlev[f] >= lev){datlev4[k] = datlev4[k]^(startbit<<f);}
					}
			}
			// **************  END LEVEL 3 - 4   ************************ //
			
			result[j] = datlev4[0]; 
	}
}

